#include "hip/hip_runtime.h"
#include <iostream>
#include "effect.h"

/*This is the Cuda function, that does special Cuda things*/
__global__
void ThisIsACudaFunction(){
    //GPU work here.
};

/*This function is a "wrapper" for the Cuda function above. Cuda functions need to be called with special thingies that won't compile in a normal C++ file*/
void DynamicSounds::Wrapper::StandardFunction(){
    std::cout << "GPU IS [not] DOING THIS" << std::endl;
    
    ThisIsACudaFunction<<<1,1>>>();
};