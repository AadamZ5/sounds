#include "hip/hip_runtime.h"
#include <iostream>
#include "sounds.h"

/*This is the Cuda function, that does special Cuda things*/
__global__
void ThisIsACudaFunction(){
    
};

extern "C" {
    void Wrapper::StandardFunction(){
        std::cout << "GPU IS DOING THIS" << std::endl;
        /*This function is a "wrapper" for the Cuda function above. Cuda functions need to be called with special thingies that won't compile in a normal C++ file*/
        ThisIsACudaFunction<<<1,1>>>();
    };
}