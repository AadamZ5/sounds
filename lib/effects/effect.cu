#include "hip/hip_runtime.h"
#include <iostream>
#include "effect.h"

/*This is the Cuda function, that does special Cuda things*/
__global__
void ThisIsACudaFunction(){
    int i = 1 + 1;
};

/*This function is a "wrapper" for the Cuda function above. Cuda functions need to be called with special thingies that won't compile in a normal C++ file*/
void DynamicSounds::StandardFunction(){
    std::cout << "GPU IS [not] DOING THIS" << std::endl;
    
    ThisIsACudaFunction<<<1,1>>>();
};